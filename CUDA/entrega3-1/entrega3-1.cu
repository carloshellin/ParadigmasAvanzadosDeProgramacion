
#include <hip/hip_runtime.h>
#include <stdio.h>

// Multiplicación de matrices MxN (4x4) por bloques y memoria compartida en cuda

const int TILE_WIDTH = 2;
const int dim = 4;

__global__ void multiplicarMatriz(int (*M_d)[dim], int (*N_d)[dim],
                                  int (*R_d)[dim])
{
    __shared__ int M_ds[TILE_WIDTH][TILE_WIDTH];
    __shared__ int N_ds[TILE_WIDTH][TILE_WIDTH];
    
    // Calcula el índice de la fila del elemento valor de M
    int fila = blockIdx.y * TILE_WIDTH + threadIdx.y;
    // Calcula el índice de la fila del elemento valor de N
    int columna = blockIdx.x * TILE_WIDTH + threadIdx.x;
    
    int valor = 0;
    
    for (int m = 0; m < dim / TILE_WIDTH; m++)
    {
        // Carga colaboritva de los tiles de Md y Nd a memoria compartida
        M_ds[threadIdx.y][threadIdx.x] = M_d[fila][m * TILE_WIDTH + threadIdx.x];
        N_ds[threadIdx.y][threadIdx.x] = N_d[m * TILE_WIDTH + threadIdx.y][columna];
        __syncthreads();
        
        // Se realiza la multiplicación mucho más rápido por la memoria compartida
        for (int k = 0; k < TILE_WIDTH; k++)
        {
            valor += M_ds[threadIdx.y][k] * N_ds[k][threadIdx.x];
        }
        __syncthreads();
    }
    
    R_d[fila][columna]= valor;
}

int main(int argc, char **argv)
{
    // Declarar todas las variables
    const int longitud = dim * dim * sizeof(int);
    int M_h[dim][dim] = 
    {
        {1, 2, 3, 4},
        {5, 6, 7, 8},
        {9, 10, 11, 12},
        {13, 14, 15, 16}
    };
    int N_h[dim][dim] = 
    {
        {4, 3, 2, 1},
        {8, 7, 6, 5},
        {12, 11, 10, 9},
        {16, 15, 14, 13}
    };
    int R_h[dim][dim] = {};
    int (*M_d)[dim];
    int (*N_d)[dim];
    int (*R_d)[dim];
    
    // Asignar memoria en el dispositivo
    hipMalloc((void **) &M_d, longitud);
    hipMalloc((void **) &N_d, longitud);
    hipMalloc((void **) &R_d, longitud);
    
    // Transferir datos al dispositivo
    hipMemcpy(M_d, M_h, longitud, hipMemcpyHostToDevice);
    hipMemcpy(N_d, N_h, longitud, hipMemcpyHostToDevice);
    
    // Ejecutar kernel en el dispositivo, cuatro bloques con cuatros hilos cada uno
    dim3 bloques(dim / TILE_WIDTH, dim / TILE_WIDTH);
    dim3 hilos(TILE_WIDTH, TILE_WIDTH);
    multiplicarMatriz<<<bloques, hilos>>>(M_d, N_d, R_d);
    
    // Transferir resultados al anfitrión
    hipMemcpy(R_h, R_d, longitud, hipMemcpyDeviceToHost);
    
    // Mostrar resultados
    printf("{\n"
           "  {1, 2, 3, 4},\n"
           "  {5, 6, 7, 8},\n"
           "  {9, 10, 11, 12},\n"
           "  {13, 14, 15, 16},\n"
           "}\n"
           "*\n"
           "{\n"
           "  {4, 3, 2, 1},\n"
           "  {8, 7, 6, 5},\n"
           "  {12, 11, 10, 9},\n"
           "  {16, 15, 14, 13},\n"
           "}\n"
           "=\n"
           "{\n"
           "  {%d, %d, %d, %d},\n"
           "  {%d, %d, %d, %d},\n"
           "  {%d, %d, %d, %d},\n"
           "  {%d, %d, %d, %d},\n"
           "}\n",
           R_h[0][0], R_h[0][1], R_h[0][2], R_h[0][3],
           R_h[1][0], R_h[1][1], R_h[1][2], R_h[1][3],
           R_h[2][0], R_h[2][1], R_h[2][2], R_h[2][3],
           R_h[3][0], R_h[3][1], R_h[3][2], R_h[3][3]);
    
    // Liberar memoria del dispositivo
    hipFree(M_d);
    hipFree(N_d);
    hipFree(R_d);
    
    return 0;
}

