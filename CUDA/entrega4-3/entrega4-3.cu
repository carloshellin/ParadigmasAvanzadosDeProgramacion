
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

/*
Programa en Cuda que realiza la convolución de matrices en GPU mediante 
múltiples bloques y memoria compartida
*/

// Tamaño de la matriz A
const int dimA = 32;

// Tamaño de la matriz B
const int dimB = 3;

const int TILE_WIDTH = 16;

__global__ void girar(int (*matrizB_d)[dimB], int (*matrizBR_d)[dimB])
{
    // Kernel para girar la matriz B de 3x3 en 180º
    
    unsigned int posicionX = threadIdx.x;
    unsigned int posicionY = threadIdx.y;
    
    switch (posicionX)
    {
        case 0:
        {
            // En el caso de estar en el hilo 0, se busca la posición 2 de x
            posicionX += 2;
            break;
        }
        
        case 2:
        {
            // En el caso de estar en el hilo 2, se busca la posición 0 de x
            posicionX -= 2;
            break;
        }
    }
    
    switch (posicionY)
    {
        case 0:
        {
            // En el caso de estar en el hilo 0, se busca la posición 2 de y
            posicionY += 2;
            break;
        }
        
        case 2:
        {
            // En el caso de estar en el hilo 2, se busca la posición 0 de y
            posicionY -= 2;
            break;
        }
    }
    
    // Se realiza el giro con posiciones e hilos
    matrizBR_d[posicionY][posicionX] = matrizB_d[threadIdx.y][threadIdx.x];
}

__global__ void convolucion(int (*matrizA_d)[dimA], int (*matrizB_d)[dimB],
                            int (*matrizR_d)[dimA])
{
    __shared__ int matrizA_ds[TILE_WIDTH][TILE_WIDTH];
    __shared__ int matrizB_ds[dimB][dimB];
    
    int fila = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int columna = blockIdx.x * TILE_WIDTH + threadIdx.x;
    
    int valor1 = 0;
    int valor2 = 0;
    int resultado = 0;
    
    matrizA_ds[threadIdx.y][threadIdx.x] = matrizA_d[fila][columna];
    
    for (int i = 0; i < dimB; i++)
    {
        for (int j = 0; j < dimB; j++)
        {
            matrizB_ds[i][j] = matrizB_d[i][j];
        }
    }
    
    __syncthreads();
    
    if (threadIdx.x > 0 && threadIdx.y > 0 && threadIdx.x < (TILE_WIDTH - 1) 
        && threadIdx.y < (TILE_WIDTH - 1))
    {
        int izquierda = threadIdx.x - 1;
        int derecha = threadIdx.x + 1;
        int arriba = threadIdx.y - 1;
        int abajo = threadIdx.y + 1;
        
        resultado += matrizA_ds[arriba][izquierda] * matrizB_ds[0][0];
        resultado += matrizA_ds[arriba][threadIdx.x] * matrizB_ds[0][1];
        resultado += matrizA_ds[arriba][derecha] * matrizB_ds[0][2];
        resultado += matrizA_ds[threadIdx.y][izquierda] * matrizB_ds[1][0];
        resultado += matrizA_ds[threadIdx.y][threadIdx.x] * matrizB_ds[1][1];
        resultado += matrizA_ds[threadIdx.y][derecha] * matrizB_ds[1][2];
        resultado += matrizA_ds[abajo][izquierda] * matrizB_ds[2][0];
        resultado += matrizA_ds[abajo][threadIdx.x] * matrizB_ds[2][1];
        resultado += matrizA_ds[abajo][derecha] * matrizB_ds[2][2];
    }
    else
    {
        int izquierda = columna - 1;
        int derecha = columna + 1;
        int arriba = fila - 1;
        int abajo = fila + 1;
        
        if (arriba >= 0)
        {
            // Se multiplica los valores de arriba
            valor1 = matrizA_d[arriba][columna];
            valor2 = matrizB_d[0][1];
            
            resultado += valor1 * valor2;
            
            if (izquierda >= 0)
            {
                // Se multiplica los valores de arriba e izquierda
                valor1 = matrizA_d[arriba][izquierda];
                valor2 = matrizB_d[0][0];
                
                resultado += valor1 * valor2;
            }
            
            if (derecha < dimA)
            {
                // Se multiplica los valores de arriba y izquierda
                valor1 = matrizA_d[arriba][derecha];
                valor2 = matrizB_d[0][2];
                
                resultado += valor1 * valor2;
            }
        }
        
        if (izquierda >= 0)
        {
            // Se multiplica los valores de izquierda
            valor1 = matrizA_d[fila][izquierda];
            valor2 = matrizB_d[1][0];
            
            resultado += valor1 * valor2;
        }
        
        
        // Se multiplica los valores del centro
        valor1 = matrizA_d[fila][columna];
        valor2 = matrizB_d[1][1];
        
        resultado += valor1 * valor2;
        
        if (derecha < dimA)
        {
            // Se multiplica los valores de derecha
            valor1 = matrizA_d[fila][derecha];
            valor2 = matrizB_d[1][2];
            
            resultado += valor1 * valor2;
        }
        
        if (abajo < dimA)
        {
            // Se multiplica los valores de abajo
            valor1 = matrizA_d[abajo][columna];
            valor2 = matrizB_d[2][1];
            
            resultado += valor1 * valor2;
            
            if (izquierda >= 0)
            {
                // Se multiplica los valores de abajo e izquierda
                valor1 = matrizA_d[abajo][izquierda];
                valor2 = matrizB_d[2][0];
                
                resultado += valor1 * valor2;
            }
            
            if (derecha < dimA)
            {
                // Se multiplica los valores de abajo y derecha
                valor1 = matrizA_d[abajo][derecha];
                valor2 = matrizB_d[2][2];
                
                resultado += valor1 * valor2;
            }
        }
    }
    
    __syncthreads();
    
    // El resultado se almacena en la matriz resultante usando fila y columna
    matrizR_d[fila][columna] = resultado;
}

void pintarMatriz(int (*matriz_h)[dimA])
{
    for (int fila = 0; fila < dimA; fila++)
    {
        printf("  {%d", matriz_h[fila][0]);
        for (int columna = 1; columna < dimA; columna++)
        {
            printf(", %d", matriz_h[fila][columna]);
        }
        
        if (fila == (dimA - 1))
        {
            printf("}\n");
        }
        else
        {
            printf("},\n");
        }
    }
    printf("}\n");
}

int main(int argc, char **argv)
{
    // Declarar todas las variables
    const int longitudA = dimA * dimA * sizeof(int);
    const int longitudB = dimB * dimB * sizeof(int);
    int matrizA_h[dimA][dimA] = {};
    int matrizB_h[dimB][dimB] = 
    {
        {0, 1, 0},
        {1, 0, 1},
        {0, 1, 0}
    };
    
    // Matriz B de prueba para demostrar el giro de 180º
    /*int matrizB_h[dimB][dimB] = 
    {
        {1, 2, 3},
        {4, 5, 6},
        {7, 8, 9}
    };*/
    int matrizR_h[dimA][dimA] = {};
    int matrizBR_h[dimB][dimB] = {};
    
    int (*matrizA_d)[dimA];
    int (*matrizB_d)[dimB];
    int (*matrizR_d)[dimA];
    int (*matrizBR_d)[dimB];
    
    // Semilla para mejorar los números pseudoaleatorios
    srand((unsigned int) time(NULL));
    
    // Escribir en la memoria del anfitrión
    for (int fila = 0; fila < dimA; fila++)
    {
        for (int columna = 0; columna < dimA; columna++)
        {
            int R = rand() % 255;
            int G = rand() % 255;
            int B = rand() % 255;
            matrizA_h[fila][columna] = R + G + B;
        }
    }
    
    // Asignar memoria en el dispositivo
    hipMalloc((void **) &matrizA_d, longitudA);
    hipMalloc((void **) &matrizB_d, longitudB);
    hipMalloc((void **) &matrizR_d, longitudA);
    hipMalloc((void **) &matrizBR_d, longitudB);
    
    // Transferir datos al dispositivo
    hipMemcpy(matrizA_d, matrizA_h, longitudA, hipMemcpyHostToDevice);
    hipMemcpy(matrizB_d, matrizB_h, longitudB, hipMemcpyHostToDevice);
    
    // Ejecutar kernel de girar en el dispositivo, un bloque con nueve hilos
    dim3 bloquesB(1, 1);
    dim3 hilosB(dimB, dimB);
    girar<<<bloquesB, hilosB>>>(matrizB_d, matrizBR_d);
    
    // Transferir el resultado de girar la matriz B al anfitrión
    hipMemcpy(matrizBR_h, matrizBR_d, longitudB, hipMemcpyDeviceToHost);
    
    // Ejecutar kernel de convolución en el dispositivo, multiples bloques con 
    // TILE_WIDTH * TILEWIDTH hilos
    dim3 bloques(dimA / TILE_WIDTH, dimA / TILE_WIDTH);
    dim3 hilos(TILE_WIDTH, TILE_WIDTH);
    convolucion<<<bloques, hilos>>>(matrizA_d, matrizBR_d, matrizR_d);
    
    // Transferir resultados al anfitrión
    hipMemcpy(matrizR_h, matrizR_d, longitudA, hipMemcpyDeviceToHost);
    
    // Mostrar resultados
    printf("Matriz A\n"
           "{\n");
    pintarMatriz(matrizA_h);
    
    printf("\nMatriz B\n");
    printf("{\n"
           "  {%d, %d, %d},\n"
           "  {%d, %d, %d},\n"
           "  {%d, %d, %d},\n"
           "}\n",
           matrizBR_h[0][0], matrizBR_h[0][1], matrizBR_h[0][2],
           matrizBR_h[1][0], matrizBR_h[1][1], matrizBR_h[1][2],
           matrizBR_h[2][0], matrizBR_h[2][1], matrizBR_h[2][2]);
    
    printf("\nMatriz resultante\n"
           "{\n");
    pintarMatriz(matrizR_h);
    
    // Liberar memoria del dispositivo
    hipFree(matrizA_d);
    hipFree(matrizB_d);
    hipFree(matrizR_d);
    hipFree(matrizBR_d);
    
    return 0;
}

